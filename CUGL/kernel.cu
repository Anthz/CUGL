#include "hip/hip_runtime.h"
#include <Windows.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include "Kernel.h"

//normal .cu file with int main() etc
//also with member functions to call kernels
//kernel::doKernel(float* data)
//{
//	kernel<<<grid, block>>>(data)
//}

__global__ void Setup_Rand(hiprandState *state)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	hiprand_init(1234, gid, 0, &state[gid]);	//change seed
}

__global__ void RandomKernel(float *buffer, dim3 dimensions)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x >= dimensions.x || y >= dimensions.y)
	{
		return;
	}

	buffer[x] *= 1.05f;
}

// int main()
// {
// 	
// }

void Kernel::ExecuteKernel(float *buffer, dim3 dimensions)
{
	dim3 blockDim(18, 1, 1);
	//dim3 gridDim(ceil((float)dimensions.x / (float)blockDim.x), ceil((float)dimensions.y / (float)blockDim.y), 1);

	RandomKernel<<<1, blockDim >>>(buffer, dimensions);
	hipError_t e = hipGetLastError();
	if(e != hipSuccess)
	{
		printf("Error");
	}

}