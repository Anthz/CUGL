#include "hip/hip_runtime.h"
#include <Windows.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <cuda_gl_interop.h>
#include <math.h>
#include "Kernel.h"

//normal .cu file with int main() etc
//also with member functions to call kernels
//kernel::doKernel(float* data)
//{
//	kernel<<<grid, block>>>(data)
//}

__global__ void AddKernel(hipSurfaceObject_t tex, dim3 dimentions)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x >= dimentions.x || y >= dimentions.y)
	{
		return;
	}

	float4 element = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
<<<<<<< HEAD
	//surf2Dwrite(element, tex, x * sizeof(float4), y);	//undefined in .cu file
=======
	surf2Dwrite(element, tex, x * sizeof(float4), y, hipBoundaryModeClamp);	//undefined in .cu file
>>>>>>> origin/master
}

// int main()
// {
// 	
// }

void Kernel::ExecuteKernel(hipSurfaceObject_t tex, dim3 dimentions)
{
	dim3 blockDim(128, 128, 1);
	dim3 gridDim(ceil((float)dimentions.x / (float)blockDim.x), ceil((float)dimentions.y / (float)blockDim.y), 1);
<<<<<<< HEAD
	AddKernel << <gridDim, blockDim >> >(tex, dimentions);
=======
	AddKernel<<<gridDim, blockDim>>>(tex, dimentions);
	hipError_t e = hipGetLastError();
	if(e != hipSuccess)
	{
		printf("Error");
	}
>>>>>>> origin/master
}